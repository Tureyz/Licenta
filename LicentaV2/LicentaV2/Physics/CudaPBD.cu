#include "hip/hip_runtime.h"
#include "CudaPBD.cuh"

#include "../Dependencies/glm/glm.hpp"

#include "../Core/CubWrappers.cuh"
#include <unordered_map>
#include "../Core/CudaUtils.cuh"

#include <thrust/host_vector.h>

namespace std
{
	template <> struct equal_to<int2>
	{
		inline bool operator()(const int2 &l, const int2 &r) const
		{
			return (l.x == r.x && l.y == r.y) || (l.x == r.y && l.y == r.x);
		}
	};

	template <> struct hash<int2>
	{
		inline size_t operator()(const int2 &v) const {
			std::hash<int> hasher;
			return hasher(v.x) ^ hasher(v.y);
		}
	};
}


__device__ const float EPS = 0.00000001f;

#define PI 3.14159265359f

void CudaPBD::CudaPBD::Init(const Physics::ClothParams parentParams, const std::vector<Rendering::VertexFormat> &verts, const std::vector<unsigned int>  &indices)
{
	m_parentParams = parentParams;

	m_particleCount = m_parentParams.dims.x * m_parentParams.dims.y;



	thrust::host_vector<bool> hfv = m_parentParams.fixedVerts;

	thrust::host_vector<int> fixedVerts;

	for (int i = 0; i < hfv.size(); ++i)
	{
		if (hfv[i])
		{
			fixedVerts.push_back(i);
		}
	}

	thrust::host_vector<float> l0s(m_particleCount * fixedVerts.size());

	for (int i = 0; i < m_particleCount; ++i)
	{
		for (int j = 0; j < fixedVerts.size(); ++j)
		{
			l0s[i * fixedVerts.size() + j] = glm::distance(verts[i].m_position, verts[fixedVerts[j]].m_position) * (1.f + m_parentParams.strainLimit);
		}
	}

	m_LRAConstraints.fixedPosID = fixedVerts;
	m_LRAConstraints.fixedVertCount = fixedVerts.size();
	m_LRAConstraints.l0 = l0s;
	m_LRAConstraints.flag.resize(m_particleCount * m_LRAConstraints.fixedVertCount);

	std::unordered_map<int2, int3> edgeTriangleMap;

	//std::vector<std::vector<bool>> takenEdges(m_parentParams.dims.x, std::vector<bool>(m_parentParams.dims.y));

	thrust::host_vector<int> hsid1;
	thrust::host_vector<int> hsid2;
	thrust::host_vector<float> hsl0;

	thrust::host_vector<int> hbid1;
	thrust::host_vector<int> hbid2;
	thrust::host_vector<int> hbid3;
	thrust::host_vector<int> hbid4;
	thrust::host_vector<float> hbphi0;

	for (int i = 0; i < indices.size(); i += 3)
	{
		int triv1 = indices[i];
		int triv2 = indices[i + 1];
		int triv3 = indices[i + 2];

		int3 triangle = make_int3(triv1, triv2, triv3);

		for (int j = 0; j < 3; ++j)
		{

			int id1 = indices[i + (j % 3)];
			int id2 = indices[i + ((j + 1) % 3)];

			int2 edge = make_int2(id1, id2);
			auto found = edgeTriangleMap.find(edge);

			if (found != edgeTriangleMap.end())
			{
				CreateStaticBendConstraint((*found).second, triangle, verts, hbid1, hbid2, hbid3, hbid4, hbphi0);
				edgeTriangleMap.erase(edge);
			}
			else
			{
				edgeTriangleMap[edge] = triangle;
				CreateStaticStretchConstraint(id1, id2, verts, hsid1, hsid2, hsl0);
			}
		}
	}

	std::vector<int> counts(m_particleCount, 0);
	std::vector<int> offsets(m_particleCount, 0);
	thrust::host_vector<int> prefixSums(m_particleCount + 1, 0);

	int totalCount = 0;

	for (int i = 0; i < hsid1.size(); ++i)
	{
		totalCount += 2;
		counts[hsid1[i]]++;
		counts[hsid2[i]]++;
	}

	for (int i = 0; i < hbid1.size(); ++i)
	{
		totalCount += 4;
		counts[hbid1[i]]++;
		counts[hbid2[i]]++;
		counts[hbid3[i]]++;
		counts[hbid4[i]]++;
	}

	for (int i = 1; i < counts.size() + 1; ++i)
	{
		prefixSums[i] = prefixSums[i - 1] + counts[i - 1];
	}

	m_staticCorrectionPrefixSums = prefixSums;

	counts = std::vector<int>(m_particleCount, 0);


	thrust::host_vector<int> hsid1o(hsid1.size(), -1);
	thrust::host_vector<int> hsid2o(hsid2.size(), -1);

	for (int i = 0; i < hsid1.size(); ++i)
	{
		hsid1o[i] = prefixSums[hsid1[i]] + (counts[hsid1[i]]++);
		hsid2o[i] = prefixSums[hsid2[i]] + (counts[hsid2[i]]++);
	}

	thrust::host_vector<int> hbid1o(hbid1.size(), -1);
	thrust::host_vector<int> hbid2o(hbid2.size(), -1);
	thrust::host_vector<int> hbid3o(hbid3.size(), -1);
	thrust::host_vector<int> hbid4o(hbid4.size(), -1);

	for (int i = 0; i < hbid1.size(); ++i)
	{
		hbid1o[i] = prefixSums[hbid1[i]] + (counts[hbid1[i]]++);
		hbid2o[i] = prefixSums[hbid2[i]] + (counts[hbid2[i]]++);
		hbid3o[i] = prefixSums[hbid3[i]] + (counts[hbid3[i]]++);
		hbid4o[i] = prefixSums[hbid4[i]] + (counts[hbid4[i]]++);
	}



	//for (int i = 0; i < hsid1.size(); ++i)
	//{
	//	if (hsid1o[i] >= totalCount)
	//	{
	//		printf("hsid1o[%d] = %d\n", i, hsid1o[i]);
	//	}
	//	if (hsid2o[i] >= totalCount)
	//	{
	//		printf("hsid2o[%d] = %d\n", i, hsid2o[i]);
	//	}
	//}

	//for (int i = 0; i < hbid1.size(); ++i)
	//{
	//	if (hbid1o[i] >= totalCount)
	//	{
	//		printf("hbid1o[%d] = %d\n", i, hbid1o[i]);
	//	}
	//	if (hbid2o[i] >= totalCount)
	//	{
	//		printf("hbid2o[%d] = %d\n", i, hbid2o[i]);
	//	}
	//	if (hbid3o[i] >= totalCount)
	//	{
	//		printf("hbid3o[%d] = %d\n", i, hbid3o[i]);
	//	}
	//	if (hbid4o[i] >= totalCount)
	//	{
	//		printf("hbid4o[%d] = %d\n", i, hbid4o[i]);
	//	}
	//}

	m_stretchConstraints.id1 = hsid1;
	m_stretchConstraints.id2 = hsid2;
	m_stretchConstraints.id1Offset = hsid1o;
	m_stretchConstraints.id2Offset = hsid2o;
	m_stretchConstraints.l0 = hsl0;
	m_stretchConstraints.k = m_parentParams.kStretch;

	m_bendConstraints.id1 = hbid1;
	m_bendConstraints.id2 = hbid2;
	m_bendConstraints.id3 = hbid3;
	m_bendConstraints.id4 = hbid4;
	m_bendConstraints.id1Offset = hbid1o;
	m_bendConstraints.id2Offset = hbid2o;
	m_bendConstraints.id3Offset = hbid3o;
	m_bendConstraints.id4Offset = hbid4o;
	m_bendConstraints.phi0 = hbphi0;
	m_bendConstraints.k = m_parentParams.kBend;



	//m_staticCorrectionIDs.resize(totalCount);
	m_staticCorrectionValues.resize(totalCount);
	m_accumulatedCorrectionValues.resize(m_particleCount);
	m_accumulatedCorrectionCounts.resize(m_particleCount);
	m_aux1.resize(m_particleCount);
	m_aux2.resize(m_particleCount);


	m_groundConstraints.flag.resize(m_particleCount);
	m_groundConstraints.nc.resize(m_particleCount);
	m_groundConstraints.qc.resize(m_particleCount);

}

void CudaPBD::CudaPBD::CreateStaticStretchConstraint(const int id1, const int id2, const std::vector<Rendering::VertexFormat> &verts,
	thrust::host_vector<int> &sid1, thrust::host_vector<int> &sid2, thrust::host_vector<float> &sl0)
{
	sid1.push_back(id1);
	sid2.push_back(id2);
	sl0.push_back(glm::distance(verts[id1].m_position, verts[id2].m_position));

	//m_stretchConstraints.k.push_back(m_parentParams.kStretch);
	//m_stretchConstraints.id1Offset.push_back(-1);
	//m_stretchConstraints.id2Offset.push_back(-1);
}

void CudaPBD::CudaPBD::CreateStaticBendConstraint(const int3 tri1, const int3 tri2, const std::vector<Rendering::VertexFormat> &verts,
	thrust::host_vector<int> &bid1, thrust::host_vector<int> &bid2, thrust::host_vector<int> &bid3,
	thrust::host_vector<int> &bid4,
	thrust::host_vector<float> &bphi0)
{
	int p1 = -1, p2 = -1, p3 = -1, p4 = -1;

	int t1[3], t2[3];
	t1[0] = tri1.x;
	t1[1] = tri1.y;
	t1[2] = tri1.z;

	t2[0] = tri2.x;
	t2[1] = tri2.y;
	t2[2] = tri2.z;
	
	for (int i = 0; i < 3; ++i)
	{
		for (int j = 0; j < 3; ++j)
		{
			if (t1[i] == t2[j])
			{
				p1 = t1[i];
				break;
			}
		}
	}

	for (int i = 0; i < 3; ++i)
	{
		for (int j = 0; j < 3; ++j)
		{
			if (t1[i] == t2[j] && t1[i] != p1)
			{
				p2 = t1[i];
				break;
			}
		}
	}


	for (int i = 0; i < 3; ++i)
	{
		if (t1[i] != p1 && t1[i] != p2)
		{
			p3 = t1[i];
			break;
		}
	}

	for (int i = 0; i < 3; ++i)
	{
		if (t2[i] != p1 && t2[i] != p2)
		{
			p4 = t2[i];
			break;
		}
	}

	glm::vec3 p21Vec = verts[p2].m_position - verts[p1].m_position;
	glm::vec3 p31Vec = verts[p3].m_position - verts[p1].m_position;
	glm::vec3 p41Vec = verts[p4].m_position - verts[p1].m_position;

	bid1.push_back(p1);
	bid2.push_back(p2);
	bid3.push_back(p3);
	bid4.push_back(p4);



	//printf("%f\n", glm::dot(glm::normalize(glm::cross(p21Vec, p31Vec)), glm::normalize(glm::cross(p21Vec, p41Vec))));
	/*m_bendConstraints.id1Offset.push_back(-1);
	m_bendConstraints.id2Offset.push_back(-1);
	m_bendConstraints.id3Offset.push_back(-1);
	m_bendConstraints.id4Offset.push_back(-1);*/

	//m_bendConstraints.k.push_back(m_parentParams.kBend);
	float phi0 = std::acos(glm::dot(glm::normalize(glm::cross(p21Vec, p31Vec)), glm::normalize(glm::cross(p21Vec, p41Vec))));

	bphi0.push_back(PI);

}

void CudaPBD::CudaPBD::CreateGroundConstraints(const thrust::device_vector<float3>& positions,
	const thrust::device_vector<float3>& prevPositions, const thrust::device_vector<float>& invMasses)
{
	const int particleCount = positions.size();

	const int numBlocks = (particleCount + CudaUtils::THREADS_PER_BLOCK - 1) / CudaUtils::THREADS_PER_BLOCK;

	_CreateGroundConstraints << <numBlocks, CudaUtils::THREADS_PER_BLOCK >> > (particleCount, cu::raw(positions), cu::raw(prevPositions), cu::raw(invMasses), m_parentParams.worldMin.y,
		m_parentParams.thickness, cu::raw(m_groundConstraints.qc), cu::raw(m_groundConstraints.nc), cu::raw(m_groundConstraints.flag));
}

void CudaPBD::CudaPBD::CreateLRAConstraints(const thrust::device_vector<float3>& positions)
{
	
	const int numBlocks = (m_particleCount * m_LRAConstraints.fixedVertCount + CudaUtils::THREADS_PER_BLOCK - 1) / CudaUtils::THREADS_PER_BLOCK;

	_CreateLRAConstraints << <numBlocks, CudaUtils::THREADS_PER_BLOCK >> > (m_particleCount, m_LRAConstraints.fixedVertCount, cu::raw(positions), cu::raw(m_LRAConstraints.fixedPosID),
		cu::raw(m_LRAConstraints.l0), cu::raw(m_LRAConstraints.flag));
}



void CudaPBD::CudaPBD::DampVelocities(const thrust::device_vector<float3>& positions, const thrust::device_vector<float>& masses,
	thrust::device_vector<float3>& velocities, void *& tempStorage, uint64_t &tempStorageSize)
{

	const int particleCount = (const int)positions.size();
	const int numBlocks = (particleCount + CudaUtils::THREADS_PER_BLOCK - 1) / CudaUtils::THREADS_PER_BLOCK;


	float massSum = CubWrap::ReduceSum(masses, tempStorage, tempStorageSize);

	float vertexMass;

	hipMemcpy(&vertexMass, cu::raw(masses), sizeof(float), hipMemcpyDeviceToHost);

	//const float massSum = vertexMass * masses.size();


	ComputeXMVMs << <numBlocks, CudaUtils::THREADS_PER_BLOCK >> > (particleCount, cu::raw(positions), cu::raw(velocities), cu::raw(masses), cu::raw(m_aux1), cu::raw(m_aux2));
	cudaCheckError();
	const float3 ximiSum = CubWrap::ReduceSum(m_aux1, tempStorage, tempStorageSize);
	const float3 vimiSum = CubWrap::ReduceSum(m_aux2, tempStorage, tempStorageSize);

	const float3 xcm = ximiSum / massSum;
	const float3 vcm = vimiSum / massSum;


	ComputeL << <numBlocks, CudaUtils::THREADS_PER_BLOCK >> > (particleCount, cu::raw(positions), xcm, cu::raw(m_aux2), cu::raw(m_aux1));
	cudaCheckError();
	const float3 L = CubWrap::ReduceSum(m_aux1, tempStorage, tempStorageSize);

	ComputeIs << <numBlocks, CudaUtils::THREADS_PER_BLOCK >> > (particleCount, cu::raw(positions), xcm, cu::raw(masses), cu::raw(m_aux1), cu::raw(m_aux2));
	cudaCheckError();
	const float3 i1 = CubWrap::ReduceSum(m_aux1, tempStorage, tempStorageSize);
	const float3 i2 = CubWrap::ReduceSum(m_aux2, tempStorage, tempStorageSize);

	const glm::mat3 I(glm::vec3(i1.y + i1.z, i2.x, i2.y),
		glm::vec3(i2.x, i1.x + i1.z, i2.z),
		glm::vec3(i2.y, i2.z, i1.x + i1.y));


	glm::vec3 omegaGLM = glm::inverse(I) * CudaUtils::MakeVec(L);

	const float3 omega = make_float3(omegaGLM.x, omegaGLM.y, omegaGLM.z);

	UpdateVels << <numBlocks, CudaUtils::THREADS_PER_BLOCK >> > (particleCount, cu::raw(positions), omega, xcm, vcm, m_parentParams.kDamp, cu::raw(velocities));
	cudaCheckError();
}

void CudaPBD::CudaPBD::PBDStepExternal(thrust::device_vector<float3> &positions, thrust::device_vector<float3> &prevPositions, const thrust::device_vector<float> &masses,
	thrust::device_vector<float3> &velocities, void *&tempStorage, uint64_t &tempStorageSize)
{
	const int particleCount = (int) positions.size();
	const int numBlocks = (particleCount + CudaUtils::THREADS_PER_BLOCK - 1) / CudaUtils::THREADS_PER_BLOCK;

	ApplyExternalForces << <numBlocks, CudaUtils::THREADS_PER_BLOCK >> > (particleCount, cu::raw(velocities), m_parentParams.gravity, m_parentParams.timestep);
	cudaCheckError();
	DampVelocities(positions, masses, velocities, tempStorage, tempStorageSize);

	ExplicitEuler << <numBlocks, CudaUtils::THREADS_PER_BLOCK >> > (particleCount, cu::raw(prevPositions), cu::raw(velocities), cu::raw(positions), m_parentParams.timestep);
	cudaCheckError();
}

void CudaPBD::CudaPBD::PBDStepSolver(thrust::device_vector<float3>& positions, const thrust::device_vector<float3>& prevPositions,
	const thrust::device_vector<float>& invMasses,
	const thrust::device_vector<Physics::PrimitiveContact>& vfContacts, const uint64_t & vfContactsSize,
	const thrust::device_vector<Physics::PrimitiveContact>& eeContacts, const uint64_t & eeContactsSize,
	void *& tempStorage, uint64_t & tempStorageSize)
{
	hipMemset(cu::raw(m_groundConstraints.flag), 0, m_groundConstraints.flag.size() * sizeof(bool));
	hipMemset(cu::raw(m_LRAConstraints.flag), 0, m_LRAConstraints.flag.size() * sizeof(bool));

	CreateGroundConstraints(positions, prevPositions, invMasses);
	CreateLRAConstraints(positions);

	for (int i = 0; i < m_parentParams.solverIterations; ++i)
	{
		hipMemset(cu::raw(m_accumulatedCorrectionCounts), 0, m_accumulatedCorrectionCounts.size() * sizeof(int));
		hipMemset(cu::raw(m_accumulatedCorrectionValues), 0, m_accumulatedCorrectionValues.size() * sizeof(float3));

		ProjectGroundConstraints(positions, invMasses);
		ProjectLRAConstraints(positions, invMasses);
		ProjectStaticConstraints(positions, invMasses, m_parentParams.solverIterations);
		ProjectDynamicConstraints(positions, invMasses, m_parentParams.thickness, vfContacts, vfContactsSize, eeContacts, eeContactsSize, tempStorage, tempStorageSize);
		ApplyCorrections(positions);
	}
}

void CudaPBD::CudaPBD::PBDStepFinal(thrust::device_vector<float3>& positions,
	thrust::device_vector<float3>& prevPositions, thrust::device_vector<float3>& velocities,
	const thrust::device_vector<bool>& fixedVerts)
{
	FinalUpdate(positions, prevPositions, velocities, fixedVerts);
}

void CudaPBD::CudaPBD::ProjectDynamicConstraints(const thrust::device_vector<float3> &positions, const thrust::device_vector<float> &invMasses,
	const float thickness,
	const thrust::device_vector<Physics::PrimitiveContact>& vfContacts,
	const uint64_t vfContactsSize,
	const thrust::device_vector<Physics::PrimitiveContact>& eeContacts,
	const uint64_t eeContactsSize,
	void *& tempStorage, uint64_t & tempStorageSize)
{
	int totalVF = vfContactsSize * 4;
	int totalEE = eeContactsSize * 4;

	int totalCorrections = totalVF + totalEE;

	if (totalCorrections == 0)
		return;

	if (m_dynamicCorrectionsSize < totalCorrections)
	{
		m_dbDynamicCorrectionID.buffers[0].resize(totalCorrections);
		m_dbDynamicCorrectionID.buffers[1].resize(totalCorrections);
		m_dbDynamicCorrectionValues.buffers[0].resize(totalCorrections);
		m_dbDynamicCorrectionValues.buffers[1].resize(totalCorrections);
		m_dDynamicCorrectionRLEUniques.resize(totalCorrections);
		m_dDynamicCorrectionRLECounts.resize(totalCorrections);
		m_dynamicCorrectionsSize = totalCorrections;
	}



	int numBlocks = (totalCorrections + CudaUtils::THREADS_PER_BLOCK - 1) / CudaUtils::THREADS_PER_BLOCK;

	_ProjectDynamicConstraints << <numBlocks, CudaUtils::THREADS_PER_BLOCK >> > (cu::raw(positions), cu::raw(invMasses), thickness,
		cu::raw(vfContacts), vfContactsSize,
		cu::raw(eeContacts), eeContactsSize,
		cu::raw(m_dbDynamicCorrectionID.buffers[m_dbDynamicCorrectionID.selector]),
		cu::raw(m_dbDynamicCorrectionValues.buffers[m_dbDynamicCorrectionValues.selector]));

	cudaCheckError();


	CubWrap::SortByKey(m_dbDynamicCorrectionID, m_dbDynamicCorrectionValues, totalCorrections, tempStorage, tempStorageSize);
	CubWrap::RLE(m_dbDynamicCorrectionID.buffers[m_dbDynamicCorrectionID.selector], totalCorrections, m_dDynamicCorrectionRLEUniques, m_dDynamicCorrectionRLECounts, m_dynamicCorrectionsRunCount,
		tempStorage, tempStorageSize);

	CubWrap::ExclusiveSumInPlace(m_dDynamicCorrectionRLECounts, m_dynamicCorrectionsRunCount, tempStorage, tempStorageSize);
	

	numBlocks = (m_dynamicCorrectionsRunCount + CudaUtils::THREADS_PER_BLOCK - 1) / CudaUtils::THREADS_PER_BLOCK;

	_ApplyDynamicCorrections << <numBlocks, CudaUtils::THREADS_PER_BLOCK >> > (m_dynamicCorrectionsRunCount, cu::raw(m_dDynamicCorrectionRLEUniques), cu::raw(m_dDynamicCorrectionRLECounts),
		cu::raw(m_dbDynamicCorrectionValues.buffers[m_dbDynamicCorrectionValues.selector]), totalCorrections, cu::raw(m_accumulatedCorrectionValues), cu::raw(m_accumulatedCorrectionCounts));

	cudaCheckError();

}

void CudaPBD::CudaPBD::ProjectStaticConstraints(const thrust::device_vector<float3>& positions, const thrust::device_vector<float>& invMasses, const int iteration)
{
	const int stretchCount = (int)m_stretchConstraints.id1.size();
	const int bendCount = (int)m_bendConstraints.id1.size();

	int numBlocks = (stretchCount + bendCount + CudaUtils::THREADS_PER_BLOCK - 1) / CudaUtils::THREADS_PER_BLOCK;

	_ProjectStaticConstraints << <numBlocks, CudaUtils::THREADS_PER_BLOCK >> > (stretchCount, bendCount, iteration,
		cu::raw(positions), cu::raw(invMasses),
		cu::raw(m_stretchConstraints.id1), cu::raw(m_stretchConstraints.id2), cu::raw(m_stretchConstraints.id1Offset), cu::raw(m_stretchConstraints.id2Offset),
		m_stretchConstraints.k, cu::raw(m_stretchConstraints.l0),
		cu::raw(m_bendConstraints.id1), cu::raw(m_bendConstraints.id2), cu::raw(m_bendConstraints.id3), cu::raw(m_bendConstraints.id4),
		cu::raw(m_bendConstraints.id1Offset), cu::raw(m_bendConstraints.id2Offset), cu::raw(m_bendConstraints.id3Offset), cu::raw(m_bendConstraints.id4Offset),
		m_bendConstraints.k, cu::raw(m_bendConstraints.phi0), cu::raw(m_staticCorrectionValues));

	cudaCheckError();

	const int particleCount = (int)positions.size();
	numBlocks = (particleCount + CudaUtils::THREADS_PER_BLOCK - 1) / CudaUtils::THREADS_PER_BLOCK;

	_ApplyStaticCorrections << <numBlocks, CudaUtils::THREADS_PER_BLOCK >> >(particleCount, cu::raw(m_staticCorrectionValues),
		cu::raw(m_staticCorrectionPrefixSums), cu::raw(m_accumulatedCorrectionValues), cu::raw(m_accumulatedCorrectionCounts));

	cudaCheckError();

}

void CudaPBD::CudaPBD::ProjectGroundConstraints(const thrust::device_vector<float3>& positions, const thrust::device_vector<float>& invMasses)
{
	const int particleCount = (int)positions.size();

	int numBlocks = (particleCount + CudaUtils::THREADS_PER_BLOCK - 1) / CudaUtils::THREADS_PER_BLOCK;

	_ApplyGroundCorrections << <numBlocks, CudaUtils::THREADS_PER_BLOCK >> > (particleCount, cu::raw(positions), cu::raw(invMasses),
		cu::raw(m_groundConstraints.flag), cu::raw(m_groundConstraints.qc), cu::raw(m_groundConstraints.nc), cu::raw(m_accumulatedCorrectionValues), cu::raw(m_accumulatedCorrectionCounts));
}

void CudaPBD::CudaPBD::ProjectLRAConstraints(const thrust::device_vector<float3>& positions, const thrust::device_vector<float>& invMasses)
{
	int numBlocks = (m_particleCount + CudaUtils::THREADS_PER_BLOCK - 1) / CudaUtils::THREADS_PER_BLOCK;

	_ApplyLRACorrections << <numBlocks, CudaUtils::THREADS_PER_BLOCK >> > (m_particleCount, m_LRAConstraints.fixedVertCount,
		cu::raw(positions), cu::raw(invMasses), cu::raw(m_LRAConstraints.fixedPosID), cu::raw(m_LRAConstraints.l0), cu::raw(m_LRAConstraints.flag),
		cu::raw(m_accumulatedCorrectionValues), cu::raw(m_accumulatedCorrectionCounts));
}


void CudaPBD::CudaPBD::ApplyCorrections(thrust::device_vector<float3>& positions)
{
	const int particleCount = m_accumulatedCorrectionValues.size();
	const int numBlocks = (particleCount + CudaUtils::THREADS_PER_BLOCK - 1) / CudaUtils::THREADS_PER_BLOCK;

	FinalCorrectionStep << <numBlocks, CudaUtils::THREADS_PER_BLOCK >> > (particleCount, cu::raw(m_accumulatedCorrectionValues), cu::raw(m_accumulatedCorrectionCounts), cu::raw(positions));
	cudaCheckError();

}

void CudaPBD::CudaPBD::FinalUpdate(thrust::device_vector<float3>& positions, thrust::device_vector<float3>& prevPositions, thrust::device_vector<float3>& velocities,
	const thrust::device_vector<bool> &fixedVerts)
{
	const int particleCount = (int)positions.size();
	int numBlocks = (particleCount + CudaUtils::THREADS_PER_BLOCK - 1) / CudaUtils::THREADS_PER_BLOCK;

	_FinalUpdate << <numBlocks, CudaUtils::THREADS_PER_BLOCK >> > (particleCount, cu::raw(positions), cu::raw(prevPositions), cu::raw(velocities), 
		cu::raw(fixedVerts), m_parentParams.timestep);

	cudaCheckError();

}

__global__ void CudaPBD::_ProjectStaticConstraints(const int stretchCount, const int bendCount, const int iteration,
	const float3 * __restrict__ positions, const float * __restrict__ invMasses,
	const int *__restrict__ sid1, const int *__restrict__ sid2, const int *__restrict__ sid1o, const int *__restrict__ sid2o,
	const float sk, const float *__restrict__ sl0,
	const int *__restrict__ bid1, const int *__restrict__ bid2, const int *__restrict__ bid3, const int *__restrict__ bid4,
	const int *__restrict__ bid1o, const int *__restrict__ bid2o, const int *__restrict__ bid3o, const int *__restrict__ bid4o,
	const float bk, const float *__restrict__ bphi0, float3 *__restrict__ corVals)
{
	int id = CudaUtils::MyID();
	if (id >= stretchCount + bendCount)
		return;

	if (id < stretchCount)
	{
		ProjectStaticStretchConstraint(id, iteration, positions, invMasses, sid1, sid2, sid1o, sid2o, sk, sl0, corVals);
	}
	else
	{
		ProjectStaticBendConstraint(id - stretchCount, iteration, positions, invMasses, bid1, bid2, bid3, bid4, bid1o, bid2o, bid3o, bid4o, bk, bphi0, corVals);
	}
}

__device__ void CudaPBD::ProjectStaticStretchConstraint(const int id, const int iteration, const float3 *__restrict__ positions, const float *__restrict__ invMasses,
	const int *__restrict__ sid1, const int *__restrict__ sid2, const int *__restrict__ sid1o, const int *__restrict__ sid2o,
	const float sk, const float *__restrict__ sl0, float3 *__restrict__ corVals)
{
	const float kp = 1 - powf(1 - sk, 1.f / iteration);
	const float w1 = invMasses[sid1[id]];
	const float w2 = invMasses[sid2[id]];

	const float wSum = w1 + w2;

	if (wSum == 0)
	{
		corVals[sid1o[id]] = make_float3(0.f, 0.f, 0.f);
		corVals[sid2o[id]] = make_float3(0.f, 0.f, 0.f);

		printf("[%d] Stretch wSum = 0\n", id);
		return;
	}

	const float3 diff = positions[sid1[id]] - positions[sid2[id]];
	const float len = CudaUtils::len(diff);

	if (len <= EPS)
	{
		corVals[sid1o[id]] = make_float3(0.f, 0.f, 0.f);
		corVals[sid2o[id]] = make_float3(0.f, 0.f, 0.f);
		printf("[%d] Stretch len = 0\n", id);
		return;
	}

	const float3 n = diff / len;

	const float3 tempTerm = (len - sl0[id]) * n * kp / wSum;
	
	corVals[sid1o[id]] = -w1 * tempTerm;
	corVals[sid2o[id]] = w2  * tempTerm;
}

__device__ void CudaPBD::ProjectStaticBendConstraint(const int id, const int iteration, const float3 *__restrict__ positions, const float *__restrict__ invMasses,
	const int *__restrict__ bid1, const int *__restrict__ bid2, const int *__restrict__ bid3, const int *__restrict__ bid4,
	const int *__restrict__ bid1o, const int *__restrict__ bid2o, const int *__restrict__ bid3o, const int *__restrict__ bid4o,
	const float bk, const float *__restrict__ bphi0, float3 *__restrict__ corVals)
{
	const float kp = 1.f - powf(1.f - bk, 1.f / iteration);
	const float w1 = invMasses[bid1[id]];
	const float w2 = invMasses[bid2[id]];
	const float w3 = invMasses[bid3[id]];
	const float w4 = invMasses[bid4[id]];

	const float3 p1 = positions[bid1[id]];
	const float3 p2 = positions[bid2[id]];
	const float3 p3 = positions[bid3[id]];
	const float3 p4 = positions[bid4[id]];

	const float3 p21 = p2 - p1;

	const float3 c23 = CudaUtils::cross(p2, p3);
	const float3 c24 = CudaUtils::cross(p2, p4);

	const float l23 = CudaUtils::len(c23);
	const float l24 = CudaUtils::len(c24);

	if (l23 <= EPS || l24 <= EPS)
	{
		corVals[bid1o[id]] = make_float3(0.f, 0.f, 0.f);
		corVals[bid2o[id]] = make_float3(0.f, 0.f, 0.f);
		corVals[bid3o[id]] = make_float3(0.f, 0.f, 0.f);
		corVals[bid4o[id]] = make_float3(0.f, 0.f, 0.f);

		printf("[%d] Bend l23: %f, l24: %f\n", id, l23, l24);
		return;
	}

	const float3 n1 = c23 / l23;
	const float3 n2 = c24 / l24;

	const float d = CudaUtils::clamp(CudaUtils::dot(n1, n2), -1.f, 1.f);

	const float3 q3 = (CudaUtils::cross(p2, n2) + (CudaUtils::cross(n1, p2) * d)) / l23;

	//if (CudaUtils::isZero(q3))
	//{
	//	printf("[%d] q3 zero p2: (%g, %g, %g), n2: (%g, %g, %g), n1: (%g, %g, %g)\n", id, p2.x, p2.y, p2.y, n2.x, n2.y, n2.z, n1.x, n1.y, n1.z);

	//}
	const float3 q4 = (CudaUtils::cross(p2, n1) + (CudaUtils::cross(n2, p2) * d)) / l24;

	/*if (CudaUtils::isZero(q4))
	{
		printf("[%d] q4 zero p2: (%g, %g, %g), n2: (%g, %g, %g), n1: (%g, %g, %g)\n", id, p2.x, p2.y, p2.y, n2.x, n2.y, n2.z, n1.x, n1.y, n1.z);

	}*/
	const float3 q2 = -((CudaUtils::cross(p3, n2) + (CudaUtils::cross(n1, p3) * d)) / l23) - ((CudaUtils::cross(p4, n1) + (CudaUtils::cross(n2, p4) * d)) / l24);

	/*if (CudaUtils::isZero(q2))
	{
		printf("[%d] q2 zero p3: (%g, %g, %g), n2: (%g, %g, %g), n1: (%g, %g, %g)\n", id, p3.x, p3.y, p3.y, n2.x, n2.y, n2.z, n1.x, n1.y, n1.z);

	}*/
	const float3 q1 = -q2 - q3 - q4;

	const float term = sqrtf(1.f - d * d) * (acosf(d) - bphi0[id]) * kp;

	const float l1 = CudaUtils::len(q1);
	const float l2 = CudaUtils::len(q2);
	const float l3 = CudaUtils::len(q3);
	const float l4 = CudaUtils::len(q4);

	const float scalingFactor = w1 * l1 * l1 + w2 * l2 * l2 + w3 * l3 * l3 + w4 * l4 * l4;

	if (scalingFactor <= EPS)
	{
		corVals[bid1o[id]] = make_float3(0.f, 0.f, 0.f);
		corVals[bid2o[id]] = make_float3(0.f, 0.f, 0.f);
		corVals[bid3o[id]] = make_float3(0.f, 0.f, 0.f);
		corVals[bid4o[id]] = make_float3(0.f, 0.f, 0.f);

		printf("[%d] Bend scalingFactor = 0, w1: %g, l1: %g, w2: %g, l2: %g, w3: %g, l3: %g, w4: %g, l4: %g, d = %g\n", id, w1, l1, w2, l2, w3, l3, w4, l4, d);
		return;
	}

	const float ts = term / scalingFactor;

	corVals[bid1o[id]] = -(w1 * ts) * q1;
	corVals[bid2o[id]] = -(w2 * ts) * q2;
	corVals[bid3o[id]] = -(w3 * ts) * q3;
	corVals[bid4o[id]] = -(w4 * ts) * q4;
}


__global__ void CudaPBD::_ProjectDynamicConstraints(const float3 * __restrict__ positions, const float * __restrict__ invMasses,
	const float thickness,
	const Physics::PrimitiveContact * __restrict__ vfs, const uint64_t vfSize,
	const Physics::PrimitiveContact * __restrict__ ees, const uint64_t eeSize,
	uint32_t * __restrict__ rawCorIDs, float3 * __restrict__ rawCorVals)
{
	const int id = CudaUtils::MyID();
	if (id >= vfSize + eeSize)
		return;

	if (id < vfSize)
	{
		ProjectVFConstraint(id, id * 4, positions, invMasses, thickness, vfs, rawCorIDs, rawCorVals);
	}
	else
	{
		ProjectEEConstraint(id - vfSize, id * 4, positions, invMasses, thickness, ees, rawCorIDs, rawCorVals);
	}
}

__device__ void CudaPBD::ProjectVFConstraint(const int id, const int myStart,
	const float3 * __restrict__ positions,
	const float * __restrict__ invMasses, const float thickness,
	const Physics::PrimitiveContact * __restrict__ vfs,
	uint32_t * __restrict__ rawCorIDs, float3 * __restrict__ rawCorVals)
{
	
	const Physics::PrimitiveContact myContact = vfs[id];

	const float b0 = CudaUtils::clamp(-myContact.w2, 0.f, 1.f);
	const float b1 = CudaUtils::clamp(-myContact.w3, 0.f, 1.f);
	const float b2 = CudaUtils::clamp(-myContact.w4, 0.f, 1.f);


	const float3 p = positions[myContact.v1];
	const float3 p0 = positions[myContact.v2];
	const float3 p1 = positions[myContact.v3];
	const float3 p2 = positions[myContact.v4];

	const float w = invMasses[myContact.v1];
	const float w0 = invMasses[myContact.v2];
	const float w1 = invMasses[myContact.v3];
	const float w2 = invMasses[myContact.v4];

	float3 q = p0 * b0 + p1 * b1 + p2 * b2;
	float3 n = p - q;

	float len = CudaUtils::len(n);

	n /= len;

	float c = len - thickness;

	float3 grad = n;
	float3 grad0 = -n * b0;
	float3 grad1 = -n * b1;
	float3 grad2 = -n * b2;

	rawCorIDs[myStart] = myContact.v1;
	rawCorIDs[myStart + 1] = myContact.v2;
	rawCorIDs[myStart + 2] = myContact.v3;
	rawCorIDs[myStart + 3] = myContact.v4;

	float s = w + w0 * b0 * b0 + w1 * b1 * b1 + w2 * b2 * b2;

	if (s == 0)
	{
		return;
	}

	s = c / s;

	s = c < 0 ? s : 0;

	if (s == 0)
	{
		return;
	}

	rawCorVals[myStart] = -s * w * grad;
	rawCorVals[myStart + 1] = -s * w0 * grad0;
	rawCorVals[myStart + 2] = -s * w1 * grad1;
	rawCorVals[myStart + 3] = -s * w2 * grad2;

}

__device__ void CudaPBD::ProjectEEConstraint(const int id, const int myStart,
	const float3 * __restrict__ positions,
	const float * __restrict__ invMasses, const float thickness,
	const Physics::PrimitiveContact * __restrict__ ees,
	uint32_t * __restrict__ rawCorIDs, float3 * __restrict__ rawCorVals)
{
	const Physics::PrimitiveContact myContact = ees[id];


	rawCorIDs[myStart] = myContact.v1;
	rawCorIDs[myStart + 1] = myContact.v2;
	rawCorIDs[myStart + 2] = myContact.v3;
	rawCorIDs[myStart + 3] = myContact.v4;

	const float3 p0 = positions[myContact.v1];
	const float3 p1 = positions[myContact.v2];
	const float3 p2 = positions[myContact.v3];
	const float3 p3 = positions[myContact.v4];

	const float w0 = invMasses[myContact.v1];
	const float w1 = invMasses[myContact.v2];
	const float w2 = invMasses[myContact.v3];
	const float w3 = invMasses[myContact.v4];


	const float b0 = myContact.w1;
	const float b1 = myContact.w2;
	const float b2 = -myContact.w3;
	const float b3 = -myContact.w4;

	float3 q0 = p0 * b0 + p1 * b1;
	float3 q1 = p2 * b2 + p3 * b3;

	float3 n = q0 - q1;

	float len = CudaUtils::len(n);
	n /= len;

	float c = len - thickness;

	float3 grad0 = n * b0;
	float3 grad1 = n * b1;
	float3 grad2 = -n * b2;
	float3 grad3 = -n * b3;

	float s = w0 * b0 * b0 + w1 * b1 * b1 + w2 * b2 * b2 + w3 * b3 * b3;

	if (s == 0)
		return;

	s = c / s;

	s = c < 0 ? s : 0;

	if (s == 0)
	{
		return;
	}

	rawCorVals[myStart] = -s * w0 * grad0;
	rawCorVals[myStart + 1] = -s * w1 * grad1;
	rawCorVals[myStart + 2] = -s * w2 * grad2;
	rawCorVals[myStart + 3] = -s * w3 * grad3;
}

__global__ void CudaPBD::_ApplyStaticCorrections(const int particleCount, const float3 * __restrict__ rawValues,
	const int * __restrict__ prefixSums,
	float3 * __restrict__ accumulatedValues, int * __restrict__ accumulatedCounts)
{
	const int id = CudaUtils::MyID();
	if (id >= particleCount)
		return;

	const int myStart = prefixSums[id];
	const int myEnd = prefixSums[id + 1];
	//const int count = myEnd - myStart;


	float3 acc = make_float3(0.f, 0.f, 0.f);

	for (int i = myStart; i < myEnd; ++i)
	{
		if (CudaUtils::isNan(rawValues[i]))
		{
			printf("[%d] rawValues[%d] is NaN\n", id, i);
		}
		acc = acc + rawValues[i];
	}

	//printf("[%d] acc: (%f, %f, %f)\n", id, acc.x, acc.y, acc.z);

	//positions[id] += (acc / count);
	accumulatedValues[id] += acc;
	accumulatedCounts[id] += (myEnd - myStart);
}

__global__ void CudaPBD::_ApplyDynamicCorrections(const int runCount, const uint32_t *__restrict__ RLEUniques,
	const int *__restrict__ RLEPrefixSums, const float3 *__restrict__ rawCorVals,
	const uint64_t rawCorValsSize, float3 *__restrict__ accumulatedCors, int * __restrict__ accumulatedCounts)
{
	int id = CudaUtils::MyID();
	if (id >= runCount)
		return;

	int myAccImpulseID = RLEUniques[id];

	int myStart = RLEPrefixSums[id];
	int myEnd = id == runCount - 1 ? rawCorValsSize : RLEPrefixSums[id + 1];

	float3 acc = make_float3(0.f, 0.f, 0.f);

	for (int i = myStart; i < myEnd; ++i)
	{
		acc += rawCorVals[i];
	}

	accumulatedCors[myAccImpulseID] += acc;
	accumulatedCounts[myAccImpulseID] += (myEnd - myStart);
}

__global__ void CudaPBD::_CreateGroundConstraints(const int particleCount,
	const float3 * __restrict__ positions, const float3 * __restrict__ prevPositions,
	const float * __restrict__ invMasses, const float groundHeight, const float thickness,
	float3 * __restrict__ qcs, float3 * __restrict__ ncs, bool * __restrict__ flags)
{
	int id = CudaUtils::MyID();
	if (id >= particleCount)
		return;

	float3 pos = positions[id];
	float3 prevPos = prevPositions[id];


	if (pos.y <= groundHeight + thickness)
	{
		float3 qc;
		float3 nc = make_float3(0.f, 1.f, 0.f);
		float3 p0 = make_float3(1.f, groundHeight + thickness, 1.f);

		float3 l = pos - prevPos;

		float d = CudaUtils::dot(p0 - pos, nc) / CudaUtils::dot(l, nc);

		qc = d * l + pos;

		flags[id] = true;
		qcs[id] = qc;
		ncs[id] = nc;

	}
}

__global__ void CudaPBD::_CreateLRAConstraints(const int particleCount, const int fixedCount,
	const float3 *__restrict__ positions, const int *__restrict__ fixedPosIDs, const float *__restrict__ l0s, bool *__restrict__ flags)
{
	int id = CudaUtils::MyID();
	if (id >= particleCount * fixedCount)
		return;


	float3 myPos = positions[id / fixedCount];
	float3 myFixed = positions[fixedPosIDs[id % fixedCount]];

	if (CudaUtils::distance(myPos, myFixed) > l0s[id])
	{
		flags[id] = true;
	}
}

__global__ void CudaPBD::_ApplyGroundCorrections(const int particleCount, const float3 *__restrict__ positions,
	const float *__restrict__ invMasses, const bool *__restrict__ flags, const float3 *__restrict__ qcs, float3 *__restrict__ ncs,
	float3 *__restrict__ accumulatedCors, int *__restrict__ accumulatedCounts)
{
	int id = CudaUtils::MyID();
	if (id >= particleCount)
		return;

	if (flags[id])
	{
		float3 pos = positions[id];
		float3 qc = qcs[id];
		float3 nc = ncs[id];

		float w = invMasses[id];

		float cVal = CudaUtils::dot(pos - qc, nc);

		if (cVal < 0)
		{
			float deriv = nc.x + nc.y + nc.z;

			
			accumulatedCors[id] += -(cVal / (w * deriv * deriv)) * w * nc;
			accumulatedCounts[id] += 1;
		}
	}
}

__global__ void CudaPBD::_ApplyLRACorrections(const int particleCount, const int fixedCount,
	const float3 *__restrict__ positions, const float *__restrict__ invMasses,
	const int *__restrict__ fixedIDs, const float *__restrict__ l0s, const bool *__restrict__ flags,
	float3 *__restrict__ accumulatedCors, int *__restrict__ accumulatedCounts)
{
	int id = CudaUtils::MyID();
	if (id >= particleCount)
		return;

	if (l0s[id] == 0.f)
		return;


	float3 acc = make_float3(0.f, 0.f, 0.f);
	int cnt = 0;

	float3 myPos = positions[id];

	for (int i = 0; i < fixedCount; ++i)
	{
		if (flags[id * fixedCount + i])
		{
			float3 fixedPos = positions[fixedIDs[i]];

			

			float3 diff = myPos - fixedPos;
			float len = CudaUtils::len(diff);

			float3 n = diff / len;

			acc += -(len - l0s[id + i]) * n;
			cnt++;
		}
	}

	accumulatedCors[id] += acc;
	accumulatedCounts[id] += cnt;
}

__global__ void CudaPBD::FinalCorrectionStep(const int particleCount,
	const float3 *__restrict__ accumulatedCors, const int *__restrict__ accumulatedCounts, float3 *__restrict__ positions)
{
	int id = CudaUtils::MyID();
	if (id >= particleCount)
		return;

	positions[id] += (accumulatedCors[id] / accumulatedCounts[id]);
}



__global__ void CudaPBD::_FinalUpdate(const int particleCount, float3 *__restrict__ positions, float3 *__restrict__ prevPositions,
	float3 *__restrict__ velocities, const bool * __restrict__ fixedVerts, const float timestep)
{
	int id = CudaUtils::MyID();
	if (id >= particleCount)
		return;

	if (fixedVerts[id])
	{
		velocities[id] = make_float3(0.f, 0.f, 0.f);		
		return;
	}
		

	velocities[id] = (positions[id] - prevPositions[id]) / timestep;
	prevPositions[id] = positions[id];
}

__global__ void CudaPBD::ComputeXMVMs(const int particleCount, const float3 *__restrict__ pos, const float3 * __restrict__ vel,
	const float *__restrict__ mass, float3 * __restrict__ XMs, float3 * __restrict__ VMs)
{
	int id = CudaUtils::MyID();
	if (id >= particleCount)
		return;

	const float m = mass[id];
	XMs[id] = pos[id] * m;
	VMs[id] = vel[id] * m;
}

__global__ void CudaPBD::ComputeL(const int particleCount, const float3 *__restrict__ pos, const float3 xcm, float3 * __restrict__ vimi, float3 *__restrict__ out)
{
	int id = CudaUtils::MyID();
	if (id >= particleCount)
		return;

	float3 ri = pos[id] - xcm;

	out[id] = CudaUtils::cross(ri, vimi[id]);
}

__global__ void CudaPBD::ComputeIs(const int particleCount, const float3 *__restrict__ pos, const float3 xcm, const float *__restrict__ mass, float3 *__restrict__ aux1, float3 *__restrict__ aux2)
{
	int id = CudaUtils::MyID();
	if (id >= particleCount)
		return;

	float3 ri = pos[id] - xcm;

	aux1[id] = make_float3(ri.x * ri.x, ri.y * ri.y, ri.z * ri.z) * mass[id];
	aux2[id] = -make_float3(ri.x * ri.y, ri.x * ri.z, ri.y * ri.z) * mass[id];
}

__global__ void CudaPBD::UpdateVels(const int particleCount, const float3 *__restrict__ pos, const float3 omega, const float3 xcm,
	const float3 vcm, const float dampingCoef, float3 *__restrict__ vel)
{
	int id = CudaUtils::MyID();
	if (id >= particleCount)
		return;

	float3 ri = pos[id] - xcm;

	float3 dv = vcm + CudaUtils::cross(omega, ri) - vel[id];

	vel[id] = vel[id] + dampingCoef * dv;
}

__global__ void CudaPBD::ApplyExternalForces(const int particleCount, float3 *__restrict__ vels, const float3 gravity, const float timestep)
{
	int id = CudaUtils::MyID();
	if (id >= particleCount)
		return;

	vels[id] = vels[id] + timestep * gravity;
}

__global__ void CudaPBD::ExplicitEuler(const int particleCount, const float3 *__restrict__ prevPositions, const float3 *__restrict__ velocities, float3 *__restrict__ positions, const float timestep)
{
	int id = CudaUtils::MyID();
	if (id >= particleCount)
		return;

	positions[id] = prevPositions[id] + timestep * velocities[id];
}





//void CudaPBD::CudaPBD::CreateStaticBendConstraint2(const int3 tri1, const int3 tri2, const std::vector<Rendering::VertexFormat>& verts,
//	thrust::host_vector<int>& bid1, thrust::host_vector<int>& bid2, thrust::host_vector<int>& bid3, thrust::host_vector<int>& bid4,
//	thrust::host_vector<float>& bphi0)
//{
//	int p1 = -1, p2 = -1, p3 = -1, p4 = -1;
//
//	int t1[3], t2[3];
//	t1[0] = tri1.x;
//	t1[1] = tri1.y;
//	t1[2] = tri1.z;
//
//	t2[0] = tri2.x;
//	t2[1] = tri2.y;
//	t2[2] = tri2.z;
//
//
//
//	for (int i = 0; i < 3; ++i)
//	{
//		for (int j = 0; j < 3; ++j)
//		{
//			if (t1[i] == t2[j])
//			{
//				p3 = t1[i];
//				break;
//			}
//		}
//	}
//
//	for (int i = 0; i < 3; ++i)
//	{
//		for (int j = 0; j < 3; ++j)
//		{
//			if (t1[i] == t2[j] && t1[i] != p1)
//			{
//				p4 = t1[i];
//				break;
//			}
//		}
//	}
//
//	for (int i = 0; i < 3; ++i)
//	{
//		if (t1[i] != p3 && t1[i] != p4)
//		{
//			p1 = t1[i];
//			break;
//		}
//	}
//
//	for (int i = 0; i < 3; ++i)
//	{
//		if (t2[i] != p3 && t2[i] != p4)
//		{
//			p2 = t2[i];
//			break;
//		}
//	}
//
//
//
//	bid1.push_back(p1);
//	bid2.push_back(p2);
//	bid3.push_back(p3);
//	bid4.push_back(p4);
//	bphi0.push_back(PI);
//
//}




//void CudaPBD::CudaPBD::ProjectConstraints(thrust::device_vector<float3> &positions, const thrust::device_vector<float> &invMasses,
//	const int iteration,
//	const thrust::device_vector<Physics::PrimitiveContact>& vfContacts,
//	const uint64_t & vfContactsSize,
//	const thrust::device_vector<Physics::PrimitiveContact>& eeContacts,
//	const uint64_t & eeContactsSize,
//	void *& tempStorage, uint64_t & tempStorageSize)
//{
//	
//
//
//	ProjectStaticConstraints(positions, invMasses, iteration);
//	cudaCheckError();
//
//
//
//	ProjectDynamicConstraints(positions, invMasses, m_parentParams.thickness, vfContacts, vfContactsSize, eeContacts, eeContactsSize, tempStorage, tempStorageSize);
//	cudaCheckError();
//
//	ApplyCorrections(positions);	
//}









//__device__ void CudaPBD::ProjectStaticBendConstraint2(const int id, const int iteration, const float3 *__restrict__ positions,
//	const float *__restrict__ invMasses,
//	const int *__restrict__ bid1, const int *__restrict__ bid2, const int *__restrict__ bid3, const int *__restrict__ bid4,
//	const int *__restrict__ bid1o, const int *__restrict__ bid2o, const int *__restrict__ bid3o, const int *__restrict__ bid4o,
//	const float bk, const float *__restrict__ bphi0, float3 *__restrict__ corVals)
//{
//	const float kp = 1.f - powf(1.f - bk, 1.f / iteration);
//
//	const float w0 = invMasses[bid1[id]];
//	const float w1 = invMasses[bid2[id]];
//	const float w2 = invMasses[bid3[id]];
//	const float w3 = invMasses[bid4[id]];
//
//	const float3 p0 = positions[bid1[id]];
//	const float3 p1 = positions[bid2[id]];
//	const float3 p2 = positions[bid3[id]];
//	const float3 p3 = positions[bid4[id]];
//
//
//	if (w0 == 0.0 && w1 == 0.0)
//	{
//		corVals[bid1o[id]] = make_float3(0.f, 0.f, 0.f);
//		corVals[bid2o[id]] = make_float3(0.f, 0.f, 0.f);
//		corVals[bid3o[id]] = make_float3(0.f, 0.f, 0.f);
//		corVals[bid4o[id]] = make_float3(0.f, 0.f, 0.f);
//
//		return;
//	}
//
//	float3 e = p3 - p2;
//
//	float elen = CudaUtils::len(e);
//
//	if (elen < EPS)
//	{
//		corVals[bid1o[id]] = make_float3(0.f, 0.f, 0.f);
//		corVals[bid2o[id]] = make_float3(0.f, 0.f, 0.f);
//		corVals[bid3o[id]] = make_float3(0.f, 0.f, 0.f);
//		corVals[bid4o[id]] = make_float3(0.f, 0.f, 0.f);
//
//		return;
//	}
//
//	float invElen = 1.f / elen;
//
//	float3 n1 = CudaUtils::cross(p2 - p0, p3 - p0);
//	float3 n2 = CudaUtils::cross(p3 - p1, p2 - p0);
//
//	float nl1 = CudaUtils::len(n1);
//	float nl2 = CudaUtils::len(n2);
//
//	n1 /= (nl1 * nl1);
//	n2 /= (nl2 * nl2);
//
//	float3 d0 = elen * n1;
//	float3 d1 = elen * n2;
//	float3 d2 = CudaUtils::dot(p0 - p3, e) * invElen * n1 + CudaUtils::dot(p1 - p3, e) * invElen * n2;
//	float3 d3 = CudaUtils::dot(p2 - p0, e) * invElen * n1 + CudaUtils::dot(p2 - p1, e) * invElen * n2;
//
//
//	
//	float d0len = CudaUtils::len(d0);
//	float d1len = CudaUtils::len(d1);
//	float d2len = CudaUtils::len(d2);
//	float d3len = CudaUtils::len(d3);
//
//	n1 = CudaUtils::normalize(n1);
//	n2 = CudaUtils::normalize(n2);
//
//	float dot = CudaUtils::clamp(CudaUtils::dot(n1, n2), -1.f, 1.f);
//
//	float phi = acosf(dot);
//
//	float lambda = w0 * d0len * d0len + w1 * d1len * d1len + w2 * d2len * d2len + w3 * d3len * d3len;
//
//	if (lambda == 0)
//	{
//		corVals[bid1o[id]] = make_float3(0.f, 0.f, 0.f);
//		corVals[bid2o[id]] = make_float3(0.f, 0.f, 0.f);
//		corVals[bid3o[id]] = make_float3(0.f, 0.f, 0.f);
//		corVals[bid4o[id]] = make_float3(0.f, 0.f, 0.f);
//
//		return;
//	}
//
//
//	lambda = (phi - bphi0[id]) / lambda * kp;
//
//	if (CudaUtils::dot(CudaUtils::cross(n1, n2), e) > 0)
//		lambda = -lambda;
//
//	corVals[bid1o[id]] = -(w0 * lambda) * d0;
//	corVals[bid2o[id]] = -(w1 * lambda) * d1;
//	corVals[bid3o[id]] = -(w2 * lambda) * d2;
//	corVals[bid4o[id]] = -(w3 * lambda) * d3;
//
//}
